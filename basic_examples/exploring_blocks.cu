
#include <hip/hip_runtime.h>
#include<stdio.h>

#define BLOCKS 3
#define GPU_THREADS_PER_BLOCK 4

__global__ void exploringBlocks()
{
    printf("Block ID = %d,  Thread ID = %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
    exploringBlocks <<<BLOCKS, GPU_THREADS_PER_BLOCK>>>();
        
    hipDeviceSynchronize();

    return 0;
}