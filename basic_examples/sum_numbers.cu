
#include <hip/hip_runtime.h>
#include<stdio.h>

#define ARRAY_SIZE 16
#define BLOCKS 1
#define VALUE_1 900
#define VALUE_2 99

__global__ void sumNumbers(const int * d_input1, const int * d_input2, int * d_output)
{
    int index = threadIdx.x;
    d_output[index] = d_input1[index] + d_input2[index];
}

int h_input1[ARRAY_SIZE];
int h_input2[ARRAY_SIZE];
int h_output[ARRAY_SIZE];

int main(void)
{
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        h_input1[i] = VALUE_1;
        h_input2[i] = VALUE_2;
    }

    int * d_input1 = NULL;
    int * d_input2 = NULL;
    int * d_output = NULL;
    
    //allocate memory on GPU
    hipMalloc( (void**) &d_input1, ARRAY_BYTES);
    hipMalloc( (void**) &d_input2, ARRAY_BYTES);
    hipMalloc( (void**) &d_output, ARRAY_BYTES);

    //transfer data to GPU
    hipMemcpy(d_input1, h_input1, ARRAY_BYTES, hipMemcpyHostToDevice);  
    hipMemcpy(d_input2, h_input2, ARRAY_BYTES, hipMemcpyHostToDevice);
    
    //launch CUDA kernel 
    sumNumbers<<<BLOCKS, ARRAY_SIZE>>>(d_input1, d_input2, d_output);   

    //transfer result back to the application data section
    hipMemcpy(h_output, d_output, ARRAY_BYTES, hipMemcpyDeviceToHost);
    
    //print the result
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        printf("%d\n", h_output[i]);
    }

    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);
    
    hipDeviceReset();           

    return 0;
}