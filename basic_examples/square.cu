
#include <hip/hip_runtime.h>
#include<stdio.h>

#define ARRAY_SIZE 64

__global__ void square (float * d_output, float * d_input)
{
    int index = threadIdx.x;
    float element = d_input[index];
    d_output[index] = element * element;
}

float h_input[ARRAY_SIZE];
float h_output[ARRAY_SIZE]; 

int main(void)
{
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

   
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        h_input[i] = float(i);
    }

    float * d_input  = NULL;
    float * d_output = NULL;
    
    //allocate memory on GPU
    hipMalloc( (void**) &d_input, ARRAY_BYTES);
    hipMalloc( (void**) &d_output, ARRAY_BYTES);

    //transfer data to GPU
    hipMemcpy(d_input, h_input, ARRAY_BYTES, hipMemcpyHostToDevice);

    square<<<1, ARRAY_SIZE>>>(d_output, d_input);

    //transfer result from GPU to the application
    hipMemcpy(h_output, d_output, ARRAY_BYTES, hipMemcpyDeviceToHost);
    
    //print the result
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        printf("%f\n", h_output[i]);
    }

    hipFree(d_input);
    hipFree(d_output);
    
    hipDeviceReset();

    return 0;
}