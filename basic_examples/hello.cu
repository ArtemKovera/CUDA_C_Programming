
#include <hip/hip_runtime.h>
#include<stdio.h>

#define BLOCKS 1
#define GPU_THREADS_PER_BLOCK 16

__global__ void helloFromGPU(void)
{
    printf("Hello from GPU!\n");
}

int main(void)
{
    printf("Hello from CPU!\n\n");

    helloFromGPU <<<BLOCKS, GPU_THREADS_PER_BLOCK>>>();
    hipDeviceReset();

    return 0;
}